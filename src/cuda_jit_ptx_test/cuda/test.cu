#include "hip/hip_runtime.h"
// #include <stdio.h>
#include <header_test.cuh>
#include <cute/tensor.hpp>

extern "C"
__global__
void
kernel() {
    using namespace cute;

    printf("hello from kernel: %d\n", func());

    auto s = make_layout(make_shape(_4{},_4{}));
    auto i = make_identity_layout(shape(s));
    print("s : "); print(s); print("\n");
    print("i : "); print(i); print("\n");
}